#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
using namespace std;
const int num_mat = 100; // total number of matrices 1 10 100 400 600 800
const int N = 4;   // square symmetric matrix dimension // 3 4
const int nTPB = 256;  // threads per block


double a[num_mat][N * N];
__host__ __device__
void r8mat_diag_get_vector(int n, double a[], double v[])
{
    int i;
    for (i = 0; i < n; i++) {
        v[i] = a[i + i * n];
    }
    return;
}
__host__ __device__
void r8mat_identity(int n, double a[])
{
    int i;
    int j;
    int k;
    k = 0;
    for (j = 0; j < n; j++)
    {
        for (i = 0; i < n; i++)
        {
            if (i == j)
            {
                a[k++] = 1.0;
            }
            else
            {
                a[k++] = 0.0;
            }
        }
    }

    return;
}
__host__ __device__
void jacobi_eigenvalue(int n, double a[], int it_max, double v[], double d[], int& it_num, int& rot_num)
{
    double* bw, * zw;
    double c, g, gapq, h;
    int i, j, k, l, m, p, q;
    double s, t, tau, term, termq, termp, theta, thresh, w;
    r8mat_identity(n, v);
    r8mat_diag_get_vector(n, a, d);
    bw = new double[n];
    zw = new double[n];
    for (i = 0; i < n; i++) {
        bw[i] = d[i];
        zw[i] = 0.0;
    }
    it_num = 0;
    rot_num = 0;
    while (it_num < it_max) {
        it_num = it_num + 1;
        //
        // The convergence threshold is based on the size of the elements in
        // the strict upper triangle of the matrix.
        //
        thresh = 0.0;
        for (i = 0; i < n; i++) {
            for (j = 0; j < i; j++) {
                thresh = thresh + a[j + i * n] * a[j + i * n];
            }
        }
        thresh = sqrt(thresh) / (double)(4 * n);
        if (thresh == 0.0) {
            break;
        }
        for (p = 0; p < n; p++) {
            for (q = p + 1; q < n; q++) {
                gapq = 10.0 * fabs(a[p + q * n]);
                termp = gapq + fabs(d[p]);
                termq = gapq + fabs(d[q]);
                //
                // Annihilate tiny offdiagonal elements.
                //
                if (4 < it_num &&
                    termp == fabs(d[p]) && termq == fabs(d[q])) {
                    a[p + q * n] = 0.0;
                }
                //
                // Otherwise, apply a rotation.
                //
                else if (thresh <= fabs(a[p + q * n])) {
                    h = d[q] - d[p];
                    term = fabs(h) + gapq;
                    if (term == fabs(h)) {
                        t = a[p + q * n] / h;
                    }
                    else {
                        theta = 0.5 * h / a[p + q * n];
                        t = 1.0 / (fabs(theta) + sqrt(1.0 + theta * theta));
                        if (theta < 0.0) {
                            t = -t;
                        }
                    }
                    c = 1.0 / sqrt(1.0 + t * t);
                    s = t * c;
                    tau = s / (1.0 + c);
                    h = t * a[p + q * n];
                    //
                    // Accumulate corrections to diagonal elements.
                    //
                    zw[p] = zw[p] - h;
                    zw[q] = zw[q] + h;
                    d[p] = d[p] - h;
                    d[q] = d[q] + h;
                    a[p + q * n] = 0.0;
                    //
                    // Rotate, using information from the upper triangle of A only.
                    //
                    for (j = 0; j < p; j++) {
                        g = a[j + p * n];
                        h = a[j + q * n];
                        a[j + p * n] = g - s * (h + g * tau);
                        a[j + q * n] = h + s * (g - h * tau);
                    }
                    for (j = p + 1; j < q; j++) {
                        g = a[p + j * n];
                        h = a[j + q * n];
                        a[p + j * n] = g - s * (h + g * tau);
                        a[j + q * n] = h + s * (g - h * tau);
                    }
                    for (j = q + 1; j < n; j++) {
                        g = a[p + j * n];
                        h = a[q + j * n];
                        a[p + j * n] = g - s * (h + g * tau);
                        a[q + j * n] = h + s * (g - h * tau);
                    }
                    //
                    // Accumulate information in the eigenvector matrix.
                    //
                    for (j = 0; j < n; j++) {
                        g = v[j + p * n];
                        h = v[j + q * n];
                        v[j + p * n] = g - s * (h + g * tau);
                        v[j + q * n] = h + s * (g - h * tau);
                    }
                    rot_num = rot_num + 1;
                }
            }
        }
        for (i = 0; i < n; i++) {
            bw[i] = bw[i] + zw[i];
            d[i] = bw[i];
            zw[i] = 0.0;
        }
    }
    //
    // Restore upper triangle of input matrix.
    //
    for (j = 0; j < n; j++) {
        for (i = 0; i < j; i++) {
            a[i + j * n] = a[j + i * n];
        }
    }
    //
    // Ascending sort the eigenvalues and eigenvectors.
    //
    for (k = 0; k < n - 1; k++) {
        m = k;
        for (l = k + 1; l < n; l++) {
            if (d[l] < d[m]) {
                m = l;
            }
        }
        if (m != k) {
            t = d[m];
            d[m] = d[k];
            d[k] = t;
            for (i = 0; i < n; i++) {
                w = v[i + m * n];
                v[i + m * n] = v[i + k * n];
                v[i + k * n] = w;
            }
        }
    }
    delete[] bw;
    delete[] zw;
    return;
}
__global__ void je(int num_matr, int n, double* a, int it_max, double* v, double* d) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int it_num;
    int rot_num;
    if (idx < num_matr) {
        jacobi_eigenvalue(n, a + (idx * n * n), it_max, v + (idx * n * n), d + (idx * n), it_num, rot_num);
    }
}

void initialize_matrix(int mat_id, int n, double* mat, double* v) {

    for (int i = 0; i < n * n; i++) *(v + (mat_id * n * n) + i) = mat[i];
}

void print_vec(int vec_id, int n, double* d) {
    cout << "matrix " << vec_id << " eigenvalues: " << endl;
    for (int i = 0; i < n; i++) cout << i << ": " << *(d + (n * vec_id) + i) << endl;
    cout << endl;
}
int main() {
    
    clock_t start, end;
    start = clock();
    ifstream ifile;
    ifile.open("pdc_mat.txt");
    int i, j;
    for (i = 0;i < num_mat;i++)
    {
        for (j = 0;j < N * N;j++)
        {
            ifile >> a[i][j];
        }
    }
    const int heapsize = num_mat * N * sizeof(double) * 2;// doubt
    const int chunks = heapsize / (8192 * 1024) + 1;// doubt
    hipError_t cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, (8192 * 1024) * chunks);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "set device heap limit failed!");
    }
    const int max_iter = 1000;
    double* h_a, * d_a, * h_v, * d_v, * h_d, * d_d;
    h_a = (double*)malloc(num_mat * N * N * sizeof(double));
    h_v = (double*)malloc(num_mat * N * N * sizeof(double));
    h_d = (double*)malloc(num_mat * N * sizeof(double));
    hipMalloc(&d_a, num_mat * N * N * sizeof(double));
    hipMalloc(&d_v, num_mat * N * N * sizeof(double));
    hipMalloc(&d_d, num_mat * N * sizeof(double));
    memset(h_a, 0, num_mat * N * N * sizeof(double));
    memset(h_v, 0, num_mat * N * N * sizeof(double));
    memset(h_d, 0, num_mat * N * sizeof(double));

    for (int i = 0;i < num_mat;i++)
    {
        initialize_matrix(i, N, a[i], h_a);
    }
    hipMemcpy(d_a, h_a, num_mat * N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, num_mat * N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_d, h_d, num_mat * N * sizeof(double), hipMemcpyHostToDevice);
    je << <num_mat, nTPB >> > (num_mat, N, d_a, max_iter, d_v, d_d);
    hipMemcpy(h_d, d_d, num_mat * N * sizeof(double), hipMemcpyDeviceToHost);
    end = clock();

    for (int i = 0;i < num_mat;i++)
    {
        print_vec(i, N, h_d);
    }
    float time_taken = ((float)(end - start)) / CLOCKS_PER_SEC;
    ofstream exec_file;
    exec_file.open("times.txt", ios::app);
    exec_file << N << " " << time_taken << " " <<num_mat << endl;
    exec_file.close();
    printf("The time taken by CUDA version is: %f s", time_taken);

    return 0;
}